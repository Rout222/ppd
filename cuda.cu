#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string.h>
#include <fstream>
#include <time.h>
using namespace std;


// laerte, caso de erro, execute o arquivo ./codigos_gerar_graficos/cuda_bench.cu

_global_
void d_main(int num_threads, int nMaior, int centro, int* matriz){
	int aux = blockDim.x * blockIdx.x + threadIdx.x;
	int n;
	int  x, dx, y, dy, perimetro, contador;
	for (n = aux+1; n <= nMaior; n += blockDim.x * num_threads)
	{
		perimetro = (2*(2*n+1))+(2*((2*n+1)-2));
		dx = 1;
		dy = 0;
		x = n;
		y = -n+1;
		contador = (2*(n-1)+1)*(2*(n-1)+1)+1; 
		for (int i = 0; i < perimetro; i++)
		{
			matriz[(centro+y)*(2*nMaior+1) + (centro+x)] = contador++;
			if (x == y || (x < 0 && x == -y) || (x > 0 && x == 1-y)){
				aux = dy;
				dy = dx;
				dx = -aux;
			}
			x = x+dx;
			y = y+dy;
		}
	}
}

int main()
{
	int nMaior;
	int q, centro;
	ifstream myfile ("input.txt");
	myfile >> nMaior;
	myfile >> q;
	int matriz[2*nMaior+1][2*nMaior+1];
	int cortes[q][4];

    // pega todos os cortes que ele deseja fazer
	for (int i = 0; i < q; i++)
	{
		for (int j = 0; j < 4; ++j)
		{
			myfile >> cortes[i][j];
		}

	}
	myfile.close();
	q = 4;
	int dim = 2*nMaior+1;
	int* d_matriz;
	hipMalloc(&d_matriz, dim*dim*sizeof(int));

    // preenchendo a matriz com os valores da espiral, indo anel por anel
	centro = nMaior;
	matriz[centro][centro] = 1;
	hipMemcpy(d_matriz, matriz, dim*dim*sizeof(int), hipMemcpyHostToDevice);
	int num_threads = 16;
	int num_blocks  = 32;
	clock_t begin_time = clock();
	d_main<<<num_threads,num_blocks>>>(num_threads, nMaior, centro,d_matriz);
	cout <<nMaior<< ","<<  float( clock () - begin_time ) /  CLOCKS_PER_SEC << ",cuda" << ",vector" << endl;
	hipMemcpy(matriz, d_matriz, dim*dim*sizeof(int), hipMemcpyDeviceToHost);
	int sum;
	begin_time = clock();
	for (int n = 0; n < q; ++n)
	{
		sum = 0;
		for (int i = min(cortes[n][2],cortes[n][0]); i <= max(cortes[n][2],cortes[n][0]); ++i)
		{
			for (int j = min(cortes[n][1],cortes[n][3]); j <= max(cortes[n][1],cortes[n][3]); ++j)
			{
				sum += matriz[centro+j][centro+i];
			}
		}
	}
	cout <<nMaior<< ", "<<  float( clock () - begin_time ) /  CLOCKS_PER_SEC << ",cuda" << ",sum" << endl;
	hipFree(&d_matriz);
	
	return 0;
}