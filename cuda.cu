#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string.h>
#include <fstream>
#include <time.h>
using namespace std;


_global_
void d_main(int num_threads, int nMaior, int centro, int* matriz){
	int aux = blockDim.x * blockIdx.x + threadIdx.x;
	int n;
	int  x, dx, y, dy, perimetro, contador;
	for (n = aux+1; n <= nMaior; n += blockDim.x * num_threads)
	{
		perimetro = (2*(2*n+1))+(2*((2*n+1)-2));
		dx = 1;
		dy = 0;
		x = n;
		y = -n+1;
		contador = (2*(n-1)+1)*(2*(n-1)+1)+1; 
		for (int i = 0; i < perimetro; i++)
		{
			matriz[(centro+y)*(2*nMaior+1) + (centro+x)] = contador++;
			if (x == y || (x < 0 && x == -y) || (x > 0 && x == 1-y)){
				aux = dy;
				dy = dx;
				dx = -aux;
			}
			x = x+dx;
			y = y+dy;
		}
	}
}

int main()
{
	int nMaior;
	int q, centro;
	clock_t begin_time;
	for (nMaior = 1; nMaior < 360; ++nMaior)
	{
		q = 4;
		int dim = 2*nMaior+1;
		int matriz[dim][dim];
		int* d_matriz;
		hipMalloc(&d_matriz, dim*dim*sizeof(int));
		int cortes[q][4];

    // pega todos os cortes que ele deseja fazer
		for (int i = 0; i < q; i++)
		{
			cortes[i][0] = -nMaior;
			cortes[i][1] = -nMaior;
			cortes[i][2] = nMaior;
			cortes[i][3] = nMaior;
		}
    // preenchendo a matriz com os valores da espiral, indo anel por anel
		centro = nMaior;
		matriz[centro][centro] = 1;
		hipMemcpy(d_matriz, matriz, dim*dim*sizeof(int), hipMemcpyHostToDevice);
		int num_threads = 16;
		int num_blocks  = 32;
		begin_time = clock();
		d_main<<<num_threads,num_blocks>>>(num_threads, nMaior, centro,d_matriz);
		cout <<nMaior<< ","<<  float( clock () - begin_time ) /  CLOCKS_PER_SEC << ",cuda" << ",vector" << endl;
		hipMemcpy(matriz, d_matriz, dim*dim*sizeof(int), hipMemcpyDeviceToHost);
		int sum;
		begin_time = clock();
		for (int n = 0; n < q; ++n)
		{
			sum = 0;
			for (int i = min(cortes[n][2],cortes[n][0]); i <= max(cortes[n][2],cortes[n][0]); ++i)
			{
				for (int j = min(cortes[n][1],cortes[n][3]); j <= max(cortes[n][1],cortes[n][3]); ++j)
				{
					sum += matriz[centro+j][centro+i];
				}
			}
		}
		cout <<nMaior<< ", "<<  float( clock () - begin_time ) /  CLOCKS_PER_SEC << ",cuda" << ",sum" << endl;
		hipFree(&d_matriz);
	}
	return 0;
}